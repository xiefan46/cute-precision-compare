#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/types.h>

#include <cute/tensor.hpp>
#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_conversion.h>
#include <cutlass/numeric_types.h>
#include <tuple>

using namespace cute;

#define PRINT(name, content) \
    print(name);             \
    print(" : ");            \
    print(content);          \
    print("\n");

#define PRINT_TENSOR(name, content) \
    print(name);             \
    print(" : ");            \
    print_tensor(content);          \
    print("\n");


namespace config {
using namespace cute;

template <typename T_, int kHeadDim_ = 64, int BLOCK_ = 64>
struct FlashConfig {
  using T = T_;
  static constexpr int kHeadDim = kHeadDim_;
  static constexpr int BLOCK = BLOCK_;

  using mma_op = SM80_16x8x16_F32F16F16F32_TN;
  using mma_traits = MMA_Traits<mma_op>;
  using mma_atom = MMA_Atom<mma_traits>;
  static constexpr int kMmaEURepeatM = 1; // 4 -> 1
  static constexpr int kMmaEURepeatN = 1;
  static constexpr int kMmaEURepeatK = 1;

  using mma_atom_shape = mma_traits::Shape_MNK;
  static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
  static constexpr int kMmaPN = 1 * kMmaEURepeatN * get<1>(mma_atom_shape{}); // 2 -> 1
  static constexpr int kMmaPK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});

  using MMA_EU_RepeatT = decltype(make_layout(make_shape(
      Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
  using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;

  using TiledMMA =
      decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));
  static constexpr int kThreadNum = size(TiledMMA{});

};

}  // namespace config


template<typename Tensor>
__forceinline__ __device__ auto fp32_to_fp16(Tensor& src_fp32) {
    using namespace cute;
    auto dest_fp16 = make_tensor_like<half_t>(src_fp32);
    auto src_fp32x2 = recast<float2>(src_fp32);
    auto dest_fp16x2 = recast<half2>(dest_fp16);
#pragma unroll
    for (int si = 0; si < size(dest_fp16x2); si++) {
        dest_fp16x2(si) = __float22half2_rn(src_fp32x2(si));
    }
    return dest_fp16;
}

template <typename config>
__global__ void compute_kv_kernel_f32_acc(const half_t* k, const half_t* v, float* kv_out, const int B, const int H, const int N)
{
  using namespace cute;
  using TiledMMA = typename config::TiledMMA;


  constexpr int BLOCK = config::BLOCK;
  constexpr int kHeadDim = config::kHeadDim;


  const int bx = blockIdx.x;
  const int tx = threadIdx.x;
  const int bs_head_offset = bx * N * kHeadDim;
  const int num_block = (N + BLOCK - 1) / BLOCK;

  __shared__ float smem_kv[kHeadDim * kHeadDim];
  Tensor Kt = make_tensor(make_gmem_ptr<half_t>(k + bs_head_offset), make_shape(Int<kHeadDim>{}, N), make_stride(Int<1>{}, Int<kHeadDim>{})); // d x N
  Tensor Vt = make_tensor(make_gmem_ptr<half_t>(v + bs_head_offset), make_shape(Int<kHeadDim>{}, N), make_stride(Int<1>{}, Int<kHeadDim>{})); // d x N

  Tensor sKV = make_tensor(make_smem_ptr<float>(&smem_kv), make_shape(Int<kHeadDim>{}, Int<kHeadDim>{}), make_stride(Int<kHeadDim>{},Int<1>{}));

  TiledMMA mma;
  ThrMMA thr_mma = mma.get_slice(tx);

  Tensor tCsKV = thr_mma.partition_C(sKV);
  clear(tCsKV);


  if (thread0()) {
    PRINT("mma size", size(mma));
    PRINT("num_block", num_block);
  }

  for (int block_id = 0; block_id < num_block; block_id++) {

    Tensor gKt = local_tile(Kt, make_tile(Int<kHeadDim>{}, Int<BLOCK>{}), make_coord(0, block_id)); // d x BLOCK
    Tensor gVt = local_tile(Vt, make_tile(Int<kHeadDim>{}, Int<BLOCK>{}), make_coord(0, block_id)); //d x BLOCK

    Tensor tAgKt = thr_mma.partition_A(gKt);
    Tensor tArKt = thr_mma.partition_fragment_A(gKt);
    Tensor tBgVt = thr_mma.partition_B(gVt);
    Tensor tBrVt = thr_mma.partition_fragment_B(gVt);

    cute::copy(tAgKt, tArKt);
    cute::copy(tBgVt, tBrVt);

    Tensor tCrNewKV = thr_mma.partition_fragment_C(sKV);
    clear(tCrNewKV);
    cute::gemm(mma, tArKt, tBrVt, tCrNewKV);


    tCrNewKV_f16 = fp32_to_fp16(tCrNewKV);

    float one = 1.0f;

    cute::axpby(one, tCrNewKV_f16, one, tCsKV);

    __syncthreads();

    Tensor gKV = make_tensor(make_gmem_ptr<float>(kv_out + block_id * kHeadDim * kHeadDim),
                             make_shape(Int<kHeadDim>{}, Int<kHeadDim>{}), make_stride(Int<kHeadDim>{}, Int<1>{})); // d x d

    Tensor tCgKV = thr_mma.partition_C(gKV);
    // copy kv result to global
    cute::copy(tCsKV, tCgKV);

    __syncthreads();
  }

}


torch::Tensor cute_compute_kv_F32F16F16F32(torch::Tensor k, torch::Tensor v) {
    int B = k.size(0);
    int H = k.size(1);
    int N = k.size(2);
    int d = k.size(3);

    int BLOCK = 64;
    int num_block = (N + BLOCK - 1) / BLOCK;

    PRINT("num_block", num_block);

    auto kv_out = torch::zeros({num_block, d, d}, torch::TensorOptions().dtype(torch::kFloat32).device(torch::Device(torch::kCUDA, 0)));

    // only for head_dim=64
    config::FlashConfig<cute::half_t> config;
    dim3 block = config.kThreadNum;
    dim3 grid(B * H);
    auto partition_kernel = compute_kv_kernel_f32_acc<decltype(config)>;
    PRINT("grid", grid);
    PRINT("block", block);

    partition_kernel<<<grid, block>>>((cute::half_t*)k.data_ptr(), (cute::half_t*)v.data_ptr(), (float*)kv_out.data_ptr(), B, H, N);
    hipDeviceSynchronize();

    return kv_out;
}